#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void cuda_ridge_detection_kernel(float *data, float *count, int rows, int cols, float thres) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && j > 0 && i < rows - 1 && j < cols - 1) {
        if (data[i * cols + j] > thres && !isnan(data[i * cols + j])) {
            int step_i = i;
            int step_j = j;
            for (int k = 0; k < 1000; k++) {
                if (step_i == 0 || step_j == 0 || step_i == rows - 1 || step_j == cols - 1) {
                    break;
                }
                int index = 4;
                float vmax = -INFINITY;
                for (int ii = 0; ii < 3; ii++) {
                    for (int jj = 0; jj < 3; jj++) {
                        float value = data[(step_i + ii - 1) * cols + step_j + jj - 1];
                        if (value > vmax) {
                            vmax = value;
                            index = jj + 3 * ii;
                        }
                    }
                }
                if (index == 4 || vmax == data[step_i * cols + step_j] || isnan(vmax)) {
                    break;
                }
                int row = index / 3;
                int col = index % 3;
                atomicAdd(&count[(step_i - 1 + row) * cols + step_j - 1 + col], 1.0f);
                step_i = step_i - 1 + row;
                step_j = step_j - 1 + col;
            }
        }
    }
}